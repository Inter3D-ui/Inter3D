#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>
#include <thrust/scan.h>


// for details of the formulae, please see https://arxiv.org/pdf/2206.05085.pdf

template <typename scalar_t>
__global__ void prefix_sums_kernel(
    const scalar_t* __restrict__ ws,
    const scalar_t* __restrict__ wts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    scalar_t* __restrict__ ws_inclusive_scan,
    scalar_t* __restrict__ ws_exclusive_scan,
    scalar_t* __restrict__ wts_inclusive_scan,
    scalar_t* __restrict__ wts_exclusive_scan
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    // compute prefix sum of ws and ws*ts
    // [a0, a1, a2, a3, ...] -> [a0, a0+a1, a0+a1+a2, a0+a1+a2+a3, ...]
    thrust::inclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_inclusive_scan+start_idx);
    thrust::inclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_inclusive_scan+start_idx);
    // [a0, a1, a2, a3, ...] -> [0, a0, a0+a1, a0+a1+a2, ...]
    thrust::exclusive_scan(thrust::device,
                           ws+start_idx,
                           ws+start_idx+N_samples,
                           ws_exclusive_scan+start_idx);
    thrust::exclusive_scan(thrust::device,
                           wts+start_idx,
                           wts+start_idx+N_samples,
                           wts_exclusive_scan+start_idx);
}


template <typename scalar_t>
__global__ void distortion_loss_fw_kernel(
    const scalar_t* __restrict__ _loss,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> loss
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];

    loss[ray_idx] = thrust::reduce(thrust::device, 
                                   _loss+start_idx,
                                   _loss+start_idx+N_samples,
                                   (scalar_t)0);
}


std::vector<torch::Tensor> distortion_loss_fw_cu(
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto wts = ws * ts;

    auto ws_inclusive_scan = torch::zeros({N}, ws.options());
    auto ws_exclusive_scan = torch::zeros({N}, ws.options());
    auto wts_inclusive_scan = torch::zeros({N}, ws.options());
    auto wts_exclusive_scan = torch::zeros({N}, ws.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu_prefix_sums", 
    ([&] {
        prefix_sums_kernel<scalar_t><<<blocks, threads>>>(
            ws.data_ptr<scalar_t>(),
            wts.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            ws_inclusive_scan.data_ptr<scalar_t>(),
            ws_exclusive_scan.data_ptr<scalar_t>(),
            wts_inclusive_scan.data_ptr<scalar_t>(),
            wts_exclusive_scan.data_ptr<scalar_t>()
        );
    }));

    auto _loss = 2*(wts_inclusive_scan*ws_exclusive_scan-
                    ws_inclusive_scan*wts_exclusive_scan) + 1.0f/3*ws*ws*deltas;

    auto loss = torch::zeros({N_rays}, ws.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_fw_cu", 
    ([&] {
        distortion_loss_fw_kernel<scalar_t><<<blocks, threads>>>(
            _loss.data_ptr<scalar_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            loss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return {loss, ws_inclusive_scan, wts_inclusive_scan};
}


template <typename scalar_t>
__global__ void distortion_loss_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dloss,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws_inclusive_scan,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> wts_inclusive_scan,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ws,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dws
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1], N_samples = rays_a[n][2];
    const int end_idx = start_idx+N_samples-1;

    const scalar_t ws_sum = ws_inclusive_scan[end_idx];
    const scalar_t wts_sum = wts_inclusive_scan[end_idx];
    // fill in dL_dws from start_idx to end_idx
    for (int s=start_idx; s<=end_idx; s++){
        dL_dws[s] = dL_dloss[ray_idx] * 2 * (
            (s==start_idx?
                (scalar_t)0:
                (ts[s]*ws_inclusive_scan[s-1]-wts_inclusive_scan[s-1])
            ) + 
            (wts_sum-wts_inclusive_scan[s]-ts[s]*(ws_sum-ws_inclusive_scan[s]))
        );
        dL_dws[s] += dL_dloss[ray_idx] * (scalar_t)2/3*ws[s]*deltas[s];
    }
}


torch::Tensor distortion_loss_bw_cu(
    const torch::Tensor dL_dloss,
    const torch::Tensor ws_inclusive_scan,
    const torch::Tensor wts_inclusive_scan,
    const torch::Tensor ws,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a
){
    const int N_rays = rays_a.size(0), N = ws.size(0);

    auto dL_dws = torch::zeros({N}, dL_dloss.options());

    const int threads = 256, blocks = (N_rays+threads-1)/threads;

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(ws.type(), "distortion_loss_bw_cu", 
    ([&] {
        distortion_loss_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_dloss.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws_inclusive_scan.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            wts_inclusive_scan.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            dL_dws.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));

    return dL_dws;
}



// -------------------------------------------------------------------------------

template <typename scalar_t>
__global__ void ne_depth_fw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> vr_samples,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1];
    const int N_samples=vr_samples[ray_idx];
    const int end_idx = start_idx+N_samples-1;
    scalar_t T = 1.0f;
    for (int s=end_idx; s>=start_idx; s--){
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        const scalar_t w = a*T;
        depth[ray_idx] += w*ts[s];
        T *= 1.0f-a;
    }

}


torch::Tensor  ne_depth_fw_cu(
    const torch::Tensor sigmas,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor vr_samples
){
    const int N_rays = rays_a.size(0), N = sigmas.size(0);
    const int threads = 256, blocks = (N_rays+threads-1)/threads;
    auto depth = torch::zeros({N_rays}, sigmas.options());


    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "ne_depth_fw_cu",
    ([&] {
        ne_depth_fw_kernel<scalar_t><<<blocks, threads>>>(
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            vr_samples.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));


    return depth;
}



template <typename scalar_t>
__global__ void ne_depth_bw_kernel(
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_ddepth,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> depth,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> sigmas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> deltas,
    const torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> ts,
    const torch::PackedTensorAccessor64<int64_t, 2, torch::RestrictPtrTraits> rays_a,
    const torch::PackedTensorAccessor64<int64_t, 1, torch::RestrictPtrTraits> vr_samples,
    torch::PackedTensorAccessor<scalar_t, 1, torch::RestrictPtrTraits, size_t> dL_dsigmas
){
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    if (n >= rays_a.size(0)) return;

    const int ray_idx = rays_a[n][0], start_idx = rays_a[n][1];
    const int N_samples=vr_samples[ray_idx];
    const int end_idx = start_idx+N_samples-1;

    scalar_t D = depth[ray_idx];
    scalar_t T = 1.0f,d = 0.0f;

    for (int s=end_idx; s>=start_idx; s--){
        const scalar_t a = 1.0f - __expf(-sigmas[s]*deltas[s]);
        d += a*T*ts[s];
        T *= 1.0f-a;
        dL_dsigmas[s] = deltas[s] * (
            dL_ddepth[ray_idx]*(ts[s]*T-(D-d))
        );
    }

}


torch::Tensor ne_depth_bw_cu(
    const torch::Tensor dL_ddepth,
    const torch::Tensor depth,
    const torch::Tensor sigmas,
    const torch::Tensor deltas,
    const torch::Tensor ts,
    const torch::Tensor rays_a,
    const torch::Tensor vr_samples
){
    const int N_rays = rays_a.size(0), N = sigmas.size(0);
    const int threads = 256, blocks = (N_rays+threads-1)/threads;
    auto dL_dsigmas = torch::zeros({N}, sigmas.options());

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(sigmas.type(), "ne_depth_bw_cu",
    ([&] {
        ne_depth_bw_kernel<scalar_t><<<blocks, threads>>>(
            dL_ddepth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            depth.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            sigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            deltas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            ts.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>(),
            rays_a.packed_accessor64<int64_t, 2, torch::RestrictPtrTraits>(),
            vr_samples.packed_accessor64<int64_t, 1, torch::RestrictPtrTraits>(),
            dL_dsigmas.packed_accessor<scalar_t, 1, torch::RestrictPtrTraits, size_t>()
        );
    }));


    return dL_dsigmas;
}
